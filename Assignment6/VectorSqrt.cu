
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include<chrono>

using namespace std;
using namespace std::chrono;

__global__ void vectorAdditionKernel(int *D_A, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= size)
        return;
    int val = D_A[idx];
    for(int i=val/2; i>=0; i--)
    {
        if( (i * i) <= val)
        {
            D_A[idx] = i;
            break;
        }
    }
}
int main()
{
    vector<int> A;
    int size;
    cout<<"Enter the size of vector needed : ";
    cin>>size;

    for(int i=0; i<size; i++)
    {
        A.push_back( rand()% 100);
    }

    int batchSize = 256;
    int *d_A;
    vector<int> C(size);

    auto start = high_resolution_clock::now();
    
    for(int batchStart = 0; batchStart < size; batchStart += batchSize)
    {
        int currentBatchSize = min(batchSize, size - batchStart);
        hipMalloc((void **)&d_A, batchSize * sizeof(int));
        hipMemcpy(d_A, A.data() + batchStart, currentBatchSize * sizeof(int), hipMemcpyHostToDevice);

        dim3 thread(batchSize, 1, 1);
        dim3 block((size + batchSize-1)/batchSize, 1, 1);
       
        vectorAdditionKernel<<<block, thread>>>(d_A, currentBatchSize);
        hipDeviceSynchronize();
        hipMemcpy(C.data() + batchStart, d_A, currentBatchSize * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_A);
    }
    auto end = high_resolution_clock::now();
    // cout<<"Original array : "<<endl;
    // for(int element:A)
    // {
    //     cout<<element<<"\t";
    // }
    // cout<<endl;

    // cout<<"Square root array : "<<endl;
    // for(int element:C)
    // {
    //     cout<<element<<"\t";
    // }
    // cout<<endl;

    auto duration = duration_cast<milliseconds>(end- start);
    cout<<" Time taken in milliseconds : "<<duration.count()<<endl;
    return 0;
}